#include "hip/hip_runtime.h"
#include "epblas/epcudakernel.h"


__global__ void _g_vsPowx(long n, float *a, float b) {
    for (long i = blockIdx.x * blockDim.x + threadIdx.x;
         i < n;
         i += blockDim.x * gridDim.x) {
        a[i] = powf(a[i], b);
    }
}

__global__ void _g_vsScale(long n, float b, float *a) {
    for (long i = blockIdx.x * blockDim.x + threadIdx.x;
         i < n;
         i += blockDim.x * gridDim.x) {
        a[i] = a[i]/b;
    }
}



__global__ void _g_vsCos(long n, float *a, float *b) {
    for (long i = blockIdx.x * blockDim.x + threadIdx.x;
         i < n;
         i += blockDim.x * gridDim.x) {
        b[i] = cosf(a[i]);
    }
}

__global__ void _g_vsSin(long n, float *a, float *b) {
    for (long i = blockIdx.x * blockDim.x + threadIdx.x;
         i < n;
         i += blockDim.x * gridDim.x) {
        b[i] = sinf(a[i]);
    }
}

__global__ void _g_vsCosSin(long n, float *a, float *b) {
    for (long i = blockIdx.x * blockDim.x + threadIdx.x;
         i < n;
         i += blockDim.x * gridDim.x) {
        sincosf(a[i],&(b[i+n]),&(b[i]))
    }
}


eparseError_t vsScale(long n,float *x,float scaler) {
    _g_vsScale <<< 4096, 256 >>> (n, scaler,x);

    return eparseSucess;
}

eparseError_t vsPowx(long n, float *a, float b) {


    _g_vsPowx <<< 4096, 256 >>> (n, a, b);


    return eparseSucess;
}


eparseError_t vsCosSinMatrix(long nrow, long ncol, float *x, float *y) {

    /**
        todo: This loop can simply be removed and fully vectorized.
    */
    for (int i = 0; i < nrow * ncol; i += nrow) {
        //_g_vsCos <<< 4096, 256 >>> (nrow, x + i, y + 2 * i);
        //_g_vsSin <<< 4096, 256 >>> (nrow, x + i, y + 2 * i + nrow);
         _g_vsCosSin <<< 4096, 256 >>> (nrow, x + i, y + 2 * i);
    }

    return eparseSucess;
}

__global__ void _g_saxpy(long n, float change, float *a, float *b) {
    for (long i = blockIdx.x * blockDim.x + threadIdx.x;
         i < n;
         i += blockDim.x * gridDim.x) {

        b[i] += change * a[i];

    }
}

eparseError_t cuda_saxpy(long n, float change, float *x, long x_idx, float *y, long y_idx){

    _g_saxpy<<< 4096, 256 >>> (n,change,x,y);
    
    return eparseSucess;
}


__global__ void _g_vsInitx(long n, float *a, float b) {
    for (long i = blockIdx.x * blockDim.x + threadIdx.x;
         i < n;
         i += blockDim.x * gridDim.x) {
        a[i] = b;
    }

}


eparseError_t vsInitx(long n, float *a, float b) {
    _g_vsInitx <<< 4096, 256 >>> (n, a, b);

    return eparseSucess;

}

__global__ void _g_setArrayByIndex(long idx, float *a, float v) {

    a[idx] = v;

}


eparseError_t setCudaArrayByIndex(long idx, float *a, float v) {
    _g_setArrayByIndex <<< 1, 1 >>> (idx, a, v);

    return eparseSucess;

}


__global__ void _g_updateArrayByIndex(long idx, float *a, float change) {

    a[idx] += change;

}


eparseError_t updateCudaArrayByIndex(long idx, float *a, float change) {
    _g_updateArrayByIndex <<< 1, 1 >>> (idx, a, change);

    return eparseSucess;
}





